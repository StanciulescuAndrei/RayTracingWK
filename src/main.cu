#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include <iostream>

#define GLFW_INCLUDE_NONE
#include <GLFW/glfw3.h>
#include "glad/glad.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

#include "cuda_common/hip/hip_runtime_api.h"

#include <hipcub/hipcub.hpp>
#include <cub/util_allocator.cuh>


#define SCREEN_WIDTH 800
#define SCREEN_HEIGHT 600

#define BLOCK_X 16
#define BLOCK_Y 16


static void error_callback(int error, const char* description)
{
    fprintf(stderr, "Error: %s\n", description);
}

static void key_callback(GLFWwindow* window, int key, int scancode, int action, int mods)
{
    if (key == GLFW_KEY_ESCAPE && action == GLFW_PRESS)
        glfwSetWindowShouldClose(window, GLFW_TRUE);
}

void framebuffer_size_callback(GLFWwindow* window, int width, int height)
{
    glViewport(0, 0, width, height);
}

void initGLContextAndWindow(GLFWwindow** window){
    
    glfwSetErrorCallback(error_callback);
 
    if (!glfwInit())
        exit(EXIT_FAILURE);
 
    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 2);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 0);
    glfwWindowHint(GLFW_RESIZABLE, GLFW_FALSE);
 
    *window = glfwCreateWindow(SCREEN_WIDTH, SCREEN_HEIGHT, "RayTracingWK", NULL, NULL);
    if (!window)
    {
        glfwTerminate();
        exit(EXIT_FAILURE);
    }
 
    glfwSetKeyCallback(*window, key_callback);
    glfwSetFramebufferSizeCallback(*window, framebuffer_size_callback);
 
    glfwMakeContextCurrent(*window);
    gladLoadGLLoader((GLADloadproc)glfwGetProcAddress);
    glfwSwapInterval(1);
}

int main(){
    GLFWwindow* window;
    initGLContextAndWindow(&window);

    /* OpenGL configuration */
    glPixelStorei(GL_UNPACK_ALIGNMENT, 16);      // 4-byte pixel alignment

    glClearColor(0, 0, 0, 0);                   // background color
    glClearStencil(0);                          // clear stencil buffer
    glClearDepth(1.0f);                         // 0 is near, 1 is far
    glEnable(GL_BLEND);  
    glBlendFunc(GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);  

    dim3 block(BLOCK_X, BLOCK_Y, 1); // One thread per pixel!
    dim3 grid(SCREEN_WIDTH / BLOCK_X + 1, SCREEN_HEIGHT / BLOCK_Y + 1, 1);

    /* Set up resources for texture writing */
    GLuint pboId;
    GLuint texId;
    GLfloat * imageData = new GLfloat[SCREEN_HEIGHT * SCREEN_WIDTH * 4];

    struct hipGraphicsResource * cuda_pbo_resource;
    void * d_pbo_buffer = NULL;

    // Initialize the texture
    glGenTextures(1, &texId);
    glBindTexture(GL_TEXTURE_2D, texId);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP);
    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA16F, SCREEN_WIDTH, SCREEN_HEIGHT, 0, GL_RGBA, GL_FLOAT, (GLvoid*)imageData);
    glBindTexture(GL_TEXTURE_2D, 0);

    // Initialize PBO
    glGenBuffers(1, &pboId);
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, pboId);
    glBufferData(GL_PIXEL_UNPACK_BUFFER, SCREEN_HEIGHT * SCREEN_WIDTH * 4 * sizeof(float), 0, GL_STREAM_DRAW);
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0);

    // Prepare CUDA interop
    checkCudaErrors(hipMalloc(&d_pbo_buffer, 4 * SCREEN_HEIGHT * SCREEN_WIDTH * sizeof(float)));
    checkCudaErrors(hipGraphicsGLRegisterBuffer(&cuda_pbo_resource, pboId, hipGraphicsRegisterFlagsNone));
    
    /* Main program loop */
    while (!glfwWindowShouldClose(window))
    {
        /* Clear color and depth buffers */
        glClear(GL_DEPTH_BUFFER_BIT | GL_COLOR_BUFFER_BIT);

        /* Bind the texture and Pixel Buffer */
        glBindBuffer(GL_PIXEL_UNPACK_BUFFER, pboId);
        glBindTexture(GL_TEXTURE_2D, texId);
        
        glTexSubImage2D(GL_TEXTURE_2D, 0, 0, 0, SCREEN_WIDTH, SCREEN_HEIGHT, GL_RGBA, GL_FLOAT, 0);

        /* Map the OpenGL resources to a CUDA memory location */
        checkCudaErrors(hipGraphicsMapResources(1, &cuda_pbo_resource, 0));
        float4* dataPointer = nullptr;
        size_t num_bytes;
        checkCudaErrors(hipGraphicsResourceGetMappedPointer((void**)&dataPointer, &num_bytes, cuda_pbo_resource));
        assert(num_bytes >= SCREEN_HEIGHT * SCREEN_WIDTH * 4 * sizeof(float));
        assert(dataPointer != nullptr);

        /* Do the rendering here */
        


        /* Unmap the OpenGL resources */
        checkCudaErrors(hipGraphicsUnmapResources(1, &cuda_pbo_resource, 0));

        /* Draw a quad which covers the entire screen */
        glBindTexture(GL_TEXTURE_2D, texId);
        glEnable(GL_TEXTURE_2D);
        glBegin(GL_QUADS);
        glNormal3f(0, 0, 1);
        glTexCoord2f(0.0f, 0.0f);   glVertex3f(-1.0f, -1.0f, 0.0f);
        glTexCoord2f(1.0f, 0.0f);   glVertex3f( 1.0f, -1.0f, 0.0f);
        glTexCoord2f(1.0f, 1.0f);   glVertex3f( 1.0f,  1.0f, 0.0f);
        glTexCoord2f(0.0f, 1.0f);   glVertex3f(-1.0f,  1.0f, 0.0f);
        glEnd();

        /* Unbind the texture and PBO */
        glBindTexture(GL_TEXTURE_2D, 0);
        glDisable(GL_TEXTURE_2D);

        /* Swap buffers and handle GLFW events */
        glfwSwapBuffers(window);
        glfwPollEvents();
    }

    /* Unmap resources and free allocated memory */
    checkCudaErrors(hipGraphicsUnregisterResource(cuda_pbo_resource));
    glDeleteTextures(1, &texId);
    glDeleteBuffers(1, &pboId);
    hipFree(d_pbo_buffer);

    delete [] imageData;

    glfwDestroyWindow(window);
    glfwTerminate();

    return 0;
}
