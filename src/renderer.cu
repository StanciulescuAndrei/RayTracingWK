#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include "camera.h"
#include "hittable.h"
#include <glm/gtc/random.hpp>
#include "utils.cuh"

const int numSceneElements = 4;

__global__ void populateScene(HitableList** hList, Sphere ** hittableBuffer, const int numElements){
    if(threadIdx.x != 0 || blockIdx.x != 0){
        return;
    }
    
    hittableBuffer[0] = new Sphere(glm::vec3(0.0f, 0.0f, -1.0f), 0.5f);
    hittableBuffer[1] = new Sphere(glm::vec3(-1.0f, 0.0f, -1.0f), 0.5f);
    hittableBuffer[2] = new Sphere(glm::vec3(1.0f, 0.0f, -1.0f), 0.5f);
    hittableBuffer[3] = new Sphere(glm::vec3(0.0f, -100.5f, -1.0f), 100.0f);

    *hList = new HitableList(hittableBuffer, numElements);
}

const int maxDepth = 10;

__device__ glm::vec3 rayColor(const Ray& ray, HitableList ** hList, int depth, hiprandState& state){
    if(depth == maxDepth){
        return glm::vec3(0.0f);
    }

    HitRecord hitRecord;

    if(hList[0]->hit(ray, 0.001f, 1000000.0f, hitRecord)){
        glm::vec3 nextDirection = hitRecord.normal + randomPointOnSphere(state);
        nextDirection = glm::normalize(nextDirection);
        return 0.5f * rayColor(Ray(hitRecord.p, nextDirection), hList, depth + 1, state);
    }
    else{
        glm::vec3 unit_direction = glm::normalize(ray.direction());
        float a = 0.5*(unit_direction[1] + 1.0);
        return (1.0f-a)*glm::vec3(1.0f, 1.0f, 1.0f) + a*glm::vec3(0.5f, 0.7f, 1.0f);
    }
}

__global__ void render(int2 resolution, float4 * data, Camera camera, HitableList ** hList){
    uint32_t idx = blockDim.x * blockIdx.x + threadIdx.x;

    int x = idx % resolution.x; // Horizontal positioning
    int y = idx / resolution.x; // Vertical positioning

    if(y >= resolution.y) return;

    hiprandState localState;
    hiprand_init(12345, idx, 0, &localState);
    
    glm::vec3 out_color(0.0f);
    const int nMultiSamples = 16;
    Ray multiSampleRays[nMultiSamples];
    camera.getPixelMultisamplex4(x, y, multiSampleRays, nMultiSamples);
    for(int i = 0; i < nMultiSamples; i++){
        out_color += 1.0f / nMultiSamples * rayColor(multiSampleRays[i], hList, 0, localState);
    }    

    data[(resolution.y - y - 1) * resolution.x + x] = {out_color[0], out_color[1], out_color[2], 1.0f};

}